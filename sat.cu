#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include <stdlib.h>
#include <set>
#include <iostream>
#include <map>
#include <time.h>
 
 
 
 
 
 
#define NO_OF_VARIABLE 25
#define NO_OF_CLAUSE 20
#define CLAUSE_SIZE 3
#define MAX_THREAD_PER_BLOCK 512
 
 
 
using namespace std;

 void generate_random_cnf( int *,int);
 
 
 
 
 
__global__ void sat_Kernel1 (int *sat_cnf2_device,int *result_h1_device,int iteration)
{

	int sequence[5];
	int temp,i,new1,val,start,end;
	int status[25];
	
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	result_h1_device[index]=1;
	
	temp=index;
	
	for(i=4;i>=0;i--)
	{
		sequence[i]=temp%5;
		temp=temp/5;
	}
	
	for(i=0;i<25;i++)
	{
		status[i]=2;
	}
	
	start=iteration *5;
	end=start+5;
	for(i=start;i<end;i++)
	{
		new1=sat_cnf2_device[i*6+sequence[i]*2];
		val=sat_cnf2_device[i*6+sequence[i]*2+1];
		if(status[new1]+val==1)
		{
			result_h1_device[index]=0;
			break;
		}
		else
		{
			status[new1]=val;
		}
		
	}
	
}
	
	





int main(void)
{
  
  
  int *sat_cnf,*sat_cnf2,*result_h1;
  int size,size2,no_of_iteration;
  int i,j,z1,*result_h1_device;
  
  
  size_t array_size,block_size,num_blocks; 
  int *sat_cnf2_device;
  
  
  
  
  
  size=NO_OF_CLAUSE *NO_OF_VARIABLE;
  size2=NO_OF_CLAUSE * CLAUSE_SIZE * 2 ;
  sat_cnf=(int*) malloc(size * sizeof(int));
  sat_cnf2=(int*) malloc(size2 * sizeof(int));
  
  

  
  
  
  
  
  generate_random_cnf( sat_cnf,size);
  
 
  
   for(i=0;i<NO_OF_CLAUSE;i++)
  {
   
  	printf("(");
  	z1=0;
  	for(j=0;j<NO_OF_VARIABLE;j++)
  	{
  		if(sat_cnf[i*NO_OF_VARIABLE+j]==1)
  		{
  				 printf("x%d",j);
  				sat_cnf2[i*CLAUSE_SIZE * 2+2*z1]=j;
  				sat_cnf2[i*CLAUSE_SIZE * 2+2*z1+1]=1;
  				
  				z1++;
  			   
  		}
  		if(sat_cnf[i*NO_OF_VARIABLE+j]==0)
  		{
  				 printf("~x%d",j);
  				sat_cnf2[i*CLAUSE_SIZE * 2+2*z1]=j;
  				sat_cnf2[i*CLAUSE_SIZE * 2+2*z1+1]=0;
  				
  				z1++;
  		}
  	 	if(z1<CLAUSE_SIZE && sat_cnf[i*NO_OF_VARIABLE+j]!=2)
  			 printf("+");
  		
  		
  	}
   
 	printf(")");
  	if(i<NO_OF_CLAUSE-1)
  		 printf(" * ");
  }
  		
  
  
  
  
  array_size=NO_OF_CLAUSE * CLAUSE_SIZE * 2 * sizeof(int) ;
  hipMalloc((void **) &sat_cnf2_device,array_size);
  hipMemcpy(sat_cnf2_device,sat_cnf2,array_size,hipMemcpyHostToDevice);
  
  
   block_size=243;
   num_blocks=1;
   no_of_iteration=4;
   
    size= no_of_iteration*243;
    result_h1=(int*) malloc(size * sizeof(int));
    
    array_size=243*sizeof(int);
    hipMalloc((void **) &result_h1_device,array_size);
   
    dim3 dimBlock(block_size);
    dim3 dimGrid(num_blocks);
    
    
    for(i=0;i<no_of_iteration;i++)
    {
    	sat_Kernel1<<<dimGrid,dimBlock>>>(sat_cnf2_device,result_h1_device,i);
    	
    	array_size=243*sizeof(int);
    	hipMemcpy(&result_h1[i*243],result_h1_device,array_size,hipMemcpyDeviceToHost);
    }
    	
    	
    	
    for(i=0;i<no_of_iteration;i++)
  {
   		for(j=0;j<243;j++)
  		{ 	
    		printf("%d ",result_h1[i*243+j]);
    	
    	}	
    printf("\n");
   }
   
         printf("\n");
     
     for(i=0;i<NO_OF_CLAUSE;i++)
  {
   		
   		for(j=0;j<6;j=j+2)
  	{
  		printf("x%d",sat_cnf2[i*6+j]);
  		if(sat_cnf2[i*6+j+1]==0)
  			printf("'");
  		printf("+");
  		}
  		printf("\n");
  }
    
    
    return 1;
  
  
  }
  
  
  
  
void generate_random_cnf( int *sat_cnf,int size)
  {
  		int i,j,flag,k;
  		int a[CLAUSE_SIZE];
  		
  		
  		for(i=0;i<size;i++)
  		{
  			sat_cnf[i]=2;
  			
		}
		
		for(i=0;i<NO_OF_CLAUSE;i++)
  		{
  			for(j=0;j<CLAUSE_SIZE;j++)
  			{
  				flag=1;
  				while(flag)
  				{
  						a[j]=rand()%NO_OF_VARIABLE;
  						flag=0;
  					    for(k=j-1;k>=0;k--)
  					    {
  					    	if(a[j]==a[k])
  					    	{
  					    		flag=1;
  					    		break;
  					    	}
  					    }
  			   }
  					
  				if(rand()%2==0)
  					sat_cnf[i*NO_OF_VARIABLE+a[j]]=0;
  				else
  					sat_cnf[i*NO_OF_VARIABLE+a[j]]=1;
  			}
  		}
  			
  }
  				
  				
		
		
		
		
 		

  
  
  
  
  
  
  
  
