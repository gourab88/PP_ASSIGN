 //Complete 3SAT solver using CUDA

// Parallel Processing course assignment

// Author : Gourab Saha
//Contact : 9051110501

// To compile : nvcc sat.cu
 
 #include<stdio.h>
 #include<hip/hip_runtime.h>
 #include<math.h>
 #include <stdlib.h>
 #include <set>
 #include <iostream>
 #include <map>
 #include <time.h>
  
  
  
  
  
  
 #define NO_OF_VARIABLE 25
 #define NO_OF_CLAUSE 20
 #define CLAUSE_SIZE 3
 #define MAX_THREAD_PER_BLOCK 512
  
  int ind;
  
 using namespace std;
 
 
  void generate_random_cnf( int *,int);
  void  reset (int *status,int *new_status);
  void find_sequence(int *sequence,int temp);
  int find_status(int start,int end,int* sat_cnf2,int *sequence,int *status,int *new_status);
  int find_one(int *final_result,int size);
   void  print_details(int i,int j,int k,int index,int *sat_cnf2);
  
  
  
  
  
 __global__ void sat_Kernel1 (int *sat_cnf2_device,int *result_h1_device,int iteration)
 {
 
	 int sequence[5];
	 int temp,i,j,new1,val,start,end;
	 int status[25];
	 
	 unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	 result_h1_device[index]=1;
	 
	 temp=index;
	 
	 for(i=4;i>=0;i--)
	 {
		 sequence[i]=temp%3;
		 temp=temp/3;
	 }
	 
	 for(i=0;i<25;i++)
	 {
		 status[i]=2;
	 }
	 
	 start=iteration *5;
	 end=start+4;
	 for(i=start,j=0;i<end;i++,j++)
	 {
		 new1=sat_cnf2_device[i*6+sequence[j]*2];
		 val=sat_cnf2_device[i*6+sequence[j]*2+1];
		 if(status[new1]+val==1)
		 {
			 result_h1_device[index]=0;
			 break;
		 }
		 else
		 {
			 status[new1]=val;
		 }
		 
	 }
	 
	 
	 __syncthreads();
 }
	 
  
  
   
 __global__ void sat_Kernel2 (int * sat_cnf2_device,int *result_h1_device,int *status_device,int *final_result_device)
 {
  
  
  
	 int sequence[5];
	 int temp,i,j,new1,val,start,end;
	 int status[25];
	 int status2[25];
	 int flag;
	 
	 unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	 temp= result_h1_device[index];
	 
	 
   
	 
	 for(i=4;i>=0;i--)
	 {
		 sequence[i]=temp%3;
		 temp=temp/3;
	 }
	 
	 for(i=0;i<25;i++)
	 {
		 status[i]=status_device[i];
		 status2[i]=2;
	 }
	 
	 
	 
	 start=15;
	 end=19;
	 for(i=start,j=0;i<end;i++,j++)
	 {
		 new1=sat_cnf2_device[i*6+sequence[j]*2];
		 val=sat_cnf2_device[i*6+sequence[j]*2+1];
		 status2[new1]=val;
		
	  }
	  
	  flag=1;
	  for(i=0;i<25;i++)
	  {
	   if((status[i]+status2[i])==1)
	   {
	      flag=0;
	      break;
	   }
	  
	  }
	  
	  final_result_device[index]=flag;
	 
	 __syncthreads();
  
     
  
 }
 
 
 
 
 
 
 
 int main(void)
 {
   
   
   int *sat_cnf,*sat_cnf2,*result_h1;
   int size,size2,no_of_iteration;
   int i,j,k,z1,*result_h1_device,*status_device;
   
   
   size_t array_size,block_size,num_blocks; 
   int *sat_cnf2_device,*final_result_device;
   int count[4];
   int status[25];
   int *final_result;
   int p1,p2,p3,flag,final_flag;
   int sequence[5],new_status_h[25],new_status_i[25],new_status_j[25],new_status_k[25];
 
  
   
   
   
   
   
   size=NO_OF_CLAUSE *NO_OF_VARIABLE;
   size2=NO_OF_CLAUSE * CLAUSE_SIZE * 2 ;
   sat_cnf=(int*) malloc(size * sizeof(int));
   sat_cnf2=(int*) malloc(size2 * sizeof(int));
   
   
 
   
   
   
   
   
   generate_random_cnf( sat_cnf,size);
   
  printf("\nRandom generated boolean expression with 25 variables and 20 clasues(with 3 literals) in Conjunctive normal form\n\n");
   
    for(i=0;i<NO_OF_CLAUSE;i++)
   {
    
	 printf("(");
	 z1=0;
	 for(j=0;j<NO_OF_VARIABLE;j++)
	 {
		 if(sat_cnf[i*NO_OF_VARIABLE+j]==1)
		 {
				  printf("x%d",j);
				 sat_cnf2[i*CLAUSE_SIZE * 2+2*z1]=j;
				 sat_cnf2[i*CLAUSE_SIZE * 2+2*z1+1]=1;
				 
				 z1++;
			    
		 }
		 if(sat_cnf[i*NO_OF_VARIABLE+j]==0)
		 {
				  printf("~x%d",j);
				 sat_cnf2[i*CLAUSE_SIZE * 2+2*z1]=j;
				 sat_cnf2[i*CLAUSE_SIZE * 2+2*z1+1]=0;
				 
				 z1++;
		 }
		 if(z1<CLAUSE_SIZE && sat_cnf[i*NO_OF_VARIABLE+j]!=2)
			  printf("+");
		 
		 
	 }
    
	 printf(")");
	 if(i<NO_OF_CLAUSE-1)
		  printf(" * ");
   }
		 
   
   
   
   
   array_size=NO_OF_CLAUSE * CLAUSE_SIZE * 2 * sizeof(int) ;
   hipMalloc((void **) &sat_cnf2_device,array_size);
   hipMemcpy(sat_cnf2_device,sat_cnf2,array_size,hipMemcpyHostToDevice);
   
   
    block_size=243;
    num_blocks=1;
    no_of_iteration=4;
    
     size= no_of_iteration*243;
     result_h1=(int*) malloc(size * sizeof(int));
     
     array_size=243*sizeof(int);
     hipMalloc((void **) &result_h1_device,array_size);
     
    array_size=no_of_iteration*243*sizeof(int);
    hipMalloc((void **) &result_h1_device,array_size);
    
     
     
     for(i=0;i<no_of_iteration;i++)
     {
	 sat_Kernel1<<<1,block_size>>>(sat_cnf2_device,result_h1_device,i);
	 
	 array_size=243*sizeof(int);
	 hipMemcpy(&result_h1[i*243],result_h1_device,array_size,hipMemcpyDeviceToHost);
     }
	 
	 
	 
     for(i=0;i<no_of_iteration;i++)
   {
      count[i]=0;
      for(j=0;j<243;j++)
      { 	
	   if(result_h1[i*243+j]==1)
	   {
	     result_h1[i*243+count[i]]=j;
	     count[i]++;
	   }
	 
      }	
    }
    
    
    
 
    
    final_result=(int *)malloc(count[3]*sizeof(int));
    
    block_size=count[3];
    num_blocks=1;
    
     dim3 dimBlock(block_size);
     dim3 dimGrid(num_blocks);
    
    
    array_size=25*sizeof(int);
    hipMalloc((void **) &status_device,array_size);
   
    
     array_size=count[3]*sizeof(int);
     hipMalloc((void **) &final_result_device,array_size);
     
      array_size=243*sizeof(int);
      hipMemcpy(result_h1_device,&result_h1[729],array_size,hipMemcpyHostToDevice);
 
     
 
    final_flag=0;
    
     for(i=0;i<25;i++)
    {
      new_status_h[i]=2;
    }
    for(i=0;i<count[0];i++)
    {
       if(final_flag==1)
       {
	   i--;
	   break;
	  
       }
       reset(status,new_status_h);
       p1=result_h1[i];
       find_sequence(sequence,p1);
       flag=find_status(0,4,sat_cnf2,sequence,status,new_status_i);
       if(flag==0)
	continue;
       
      for(j=0;j<count[1];j++)
      {
	if(final_flag==1)
	{
	  j--;
	   break;
	}
	reset(status,new_status_i);
	p2=result_h1[243+j];
	find_sequence(sequence,p2);
	flag=find_status(5,9,sat_cnf2,sequence,status,new_status_j);
	if(flag==0)
	 continue;
	
       for(k=0;k<count[2];k++)
       {
	  reset(status,new_status_j);
	  p3=result_h1[2*243+k];
	 find_sequence(sequence,p3);
	 flag=find_status(10,14,sat_cnf2,sequence,status,new_status_k);
	 if(flag==0)
	  continue;
	 else
	 {
	  reset(status,new_status_k);
	  array_size=25*sizeof(int);
	  hipMemcpy(status_device,status,array_size,hipMemcpyHostToDevice);
	  
	  
	  sat_Kernel2<<<dimGrid,dimBlock>>>(sat_cnf2,result_h1_device,status_device,final_result_device);
	  
	  array_size=count[3]*sizeof(int);
	 hipMemcpy(final_result,final_result_device,array_size,hipMemcpyDeviceToHost);
	  printf("\n\n");
	 
	  final_flag=find_one(final_result,count[3]);
	  if(final_flag==1)
	   break;
	 }
       }
      }
    }
    
    
   
    
    
    printf("\n\n");
    
    if(final_flag==1)
    {
      printf(" \nThe CNF is satisfiable with the following assignment \n\n");
      print_details(i,j,k,ind,sat_cnf2);
    }
    
    else
    {
       printf(" \nThe CNF is not satisfiable !! \n\n");
    }
    
    
    
    
   
   /* 
      for(i=0;i<no_of_iteration;i++)
   {
     printf("\nno of 1s :%d\n",count[i]);
      for(j=0;j<243;j++)
      { 	
	   printf("%d ",result_h1[i*243+j]);
	    
	 
      }
      printf("\n");
    }
    
	  printf("\n");
	  
	  
	  
	  
	  
	  
	  
	  
	  
	  
      
      for(i=0;i<NO_OF_CLAUSE;i++)
   {
		 
		 for(j=0;j<6;j=j+2)
	 {
		 printf("x%d",sat_cnf2[i*6+j]);
		 if(sat_cnf2[i*6+j+1]==0)
			 printf("'");
		 printf("+");
		 }
		 printf("\n");
   }
     
   */
   
   
   hipFree(sat_cnf2_device);
   hipFree(final_result_device);
   hipFree(result_h1_device);
   hipFree(status_device);
   
   free(sat_cnf);
   free(sat_cnf2);
   free(result_h1);
   free(final_result);
   
     return 1;
   
   
   }
   
   
   
   
 void generate_random_cnf( int *sat_cnf,int size)
   {
		 int i,j,flag,k;
		 int a[CLAUSE_SIZE];
		 
		 
		 for(i=0;i<size;i++)
		 {
			 sat_cnf[i]=2;
			 
		 }
		 
		 for(i=0;i<NO_OF_CLAUSE;i++)
		 {
			 for(j=0;j<CLAUSE_SIZE;j++)
			 {
				 flag=1;
				 while(flag)
				 {
						 a[j]=rand()%NO_OF_VARIABLE;
						 flag=0;
					     for(k=j-1;k>=0;k--)
					     {
						 if(a[j]==a[k])
						 {
							 flag=1;
							 break;
						 }
					     }
			    }
					 
				 if(rand()%2==0)
					 sat_cnf[i*NO_OF_VARIABLE+a[j]]=0;
				 else
					 sat_cnf[i*NO_OF_VARIABLE+a[j]]=1;
			 }
		 }
			 
   }
				 
				 
   
   
   
  void  reset (int *status ,int *new_status)
   {
   
    int i;
     for(i=0;i<25;i++)
	 {
		 status[i]=new_status[i];
	 }
	 
   }
		 
  void find_sequence(int *sequence,int temp)
  {
    int i;
    for(i=4;i>=0;i--)
	 {
		 sequence[i]=temp%3;
		 temp=temp/3;
	 }
  }
  
  
  
  
  int find_status(int start,int end,int* sat_cnf2,int *sequence,int *status,int *new_status)
  {
   int i,j,new1,val;
   
   for(i=0;i<25;i++)
    new_status[i]=status[i];
   for(i=start,j=0;i<end;i++,j++)
   {
	new1=sat_cnf2[i*6+sequence[j]*2];
	val=sat_cnf2[i*6+sequence[j]*2+1];
	if(new_status[new1]+val==1)
	{
	   return 0;
	 }
	 else
	 {
	     new_status[new1]=val;
	  }
		 
   }
   return 1;
   
  }
		 
  
  
  int find_one(int *final_result,int size)
  {
    int i,flag=0;
    for(i=0;i<size;i++)
    {
     if(final_result[i]==1)
     {
      flag=1;
      ind=i;
      break;
     }
    }
    return flag;
  }
 
 
 
 void  print_details(int i,int j,int k,int index,int *sat_cnf2)
 {
   int sequence[5];
    int temp,x,new1,y,val;
    int status[25];
    
    for(x=0;x<25;x++)
     status[x]=2;
  
   
    temp=i;
   
    for(x=4;x>=0;x--)
   {
		 sequence[x]=temp%3;
		 temp=temp/3;
    }
    
    for(x=0,y=0;x<4;x++,y++)
    {
	  new1=sat_cnf2[x*6+sequence[y]*2];
	   val=sat_cnf2[x*6+sequence[y]*2+1];
	   status[new1]=val;
		
     }
     
     
     
     temp=j;
   
    for(x=4;x>=0;x--)
   {
		 sequence[x]=temp%3;
		 temp=temp/3;
    }
    
    for(x=5,y=0;x<9;x++,y++)
    {
	  new1=sat_cnf2[x*6+sequence[y]*2];
	   val=sat_cnf2[x*6+sequence[y]*2+1];
	   status[new1]=val;
		
     }
  
  
   temp=k;
   
    for(x=4;x>=0;x--)
   {
		 sequence[x]=temp%3;
		 temp=temp/3;
    }
    
    for(x=10,y=0;x<14;x++,y++)
    {
	  new1=sat_cnf2[x*6+sequence[y]*2];
	   val=sat_cnf2[x*6+sequence[y]*2+1];
	   status[new1]=val;
		
     }
     
     
     
     
     temp=i;
   
    for(x=4;x>=0;x--)
   {
		 sequence[x]=temp%3;
		 temp=temp/3;
    }
    
    for(x=15,y=0;x<19;x++,y++)
    {
	  new1=sat_cnf2[x*6+sequence[y]*2];
	   val=sat_cnf2[x*6+sequence[y]*2+1];
	   status[new1]=val;
		
     }
     
     
       
    for(x=0;x<25;x++)
    {
     if(status[x]==2)
      printf("\nx%d : Dont Care(0 or 1)",x);
     else
     printf("\nx%d : %d",x,status[x]);
    }
    
    printf("\n\n\n");
  
 }
   
   
   
   
   
   
   
   
