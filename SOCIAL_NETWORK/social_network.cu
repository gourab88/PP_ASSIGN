#include "hip/hip_runtime.h"
// Stimulation of information diffusion in social network with 
//respect to time in social network using CUDA

// Parallel Processing course assignment

// Author : Gourab Saha
//Contact : 9051110501

// To compile : nvcc prog5.0.cu 



#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include <stdlib.h>
#include <set>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <iostream>
#include <map>
#include <time.h>
 



#define NO_OF_USER 100
#define MAX_NEIGHBOUR 4
#define MIN_NEIGHBOUR 10
#define SIZE_OF_LOCAL_LIST 10
#define SIZE_OF_GLOBAL_LIST 500
#define MAX_TIMESTAMP 2880
#define LAMDA_ARRIVAL_GLOBAL 0.08
#define LAMDA_ARRIVAL_USER  0.1
#define TIMEUNIT_BEFORE_ZERO 1000
#define INITIAL_WEIGHT_GL 100
#define INITIAL_WEIGHT_LL 100
#define GLOBAL_DECAY_RATE 0.05
#define LOCAL_DECAY_RATE 0.05
#define SAMPLING_INTERVAL 6
#define WEIGHT_THRESOLD 0.01
 

using namespace std;
int no_of_topics;
int no_of_iteration;
int global_zero_index;

void make_graph(int *v_graph , int *e_graph);
void make_global_list(float * global_list);
void initialize_graph(float *global_list,float *local_list,int **topic_book_keeping);
void generate_initial_action_time(float **user_action);
 






/*
      **********************************************
      **                                          **
      **            Kernel Function               **
      **                                          **
      ********************************************** 
 */  
   
 
 

   
__global__ void  Take_Action_Kernel(int *v_graph_device,int *e_graph_device,float *local_list_device,
                                    float * global_list_device,short *active_now_device,short *active_element_ids_device,
                                    short *count_device,int *no_of_topics_device,float *current_time_device,int *result_device
                                    ,hiprandState * state, unsigned long seed)
{
      clock_t start_parallel_device= clock();
     
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
 
     
  
    
    float weights[500];
    float roulette_wheel[500];
    int current_index,count=0,i,j,n,user,topic;
    float weight,sum=0.0f,x=0.0f;
    int node_index,neighbours,last_change,N;
    
    node_index=active_element_ids_device[index];
    
 
    neighbours=v_graph_device[node_index+1]-v_graph_device[node_index];
    
    for(i=0;i<*no_of_topics_device;i++)
        weights[i]=0.0f;
    
    for(i=0;i<neighbours;i++)
    {
        current_index=v_graph_device[node_index]+i;
        j=(*no_of_topics_device)*e_graph_device[current_index];
        N=*no_of_topics_device+j;
        for(count=j;count<N;count++)
        {
            if(local_list_device[count] < -1000.0f)
                continue;
            weight=INITIAL_WEIGHT_LL*(float)expf(-1*LOCAL_DECAY_RATE * (*current_time_device-local_list_device[count]));
            if(weight<WEIGHT_THRESOLD)
                local_list_device[count]=-9999.0f;
            else
            {
               weights[count-j]=weights[count-j]+weight;
               sum=sum+weight;
            }
               
                
        }
    }
    
    for(count=0;count<*no_of_topics_device;count++)
    {
         if(global_list_device[count]>*current_time_device)
  			break;
 
        if(global_list_device[count] < -1000.0f)
                continue;
        weight=INITIAL_WEIGHT_GL*(float)expf(-1*GLOBAL_DECAY_RATE * (*current_time_device-global_list_device[count]));
        if(weight<WEIGHT_THRESOLD)
            global_list_device[count]=-9999.0f;
        else
            {
               weights[count]=weights[count]+weight;
               sum=sum+weight;
            }
    }
    
    roulette_wheel[0]=0.0f;
    x=0.0f;
     for(count=0;count<*no_of_topics_device;count++)
     {
         weights[count]=weights[count]/sum;
        roulette_wheel[count+1]=x+ weights[count];
        x=roulette_wheel[count+1]; 
     }
     
     
    
    hiprandState localState = state[threadIdx.x];
    hiprand_init ( seed,threadIdx.x, 0, &localState);
   
    x= hiprand_uniform( &localState );
   state[threadIdx.x] = localState; 
    
    last_change=0;
     i=0;
     
    while( roulette_wheel[i]<x)
    {
        if(i>0 && roulette_wheel[i]!=roulette_wheel[i-1])
        	last_change=i;
        i++;
       }
     i=last_change;
    
    __syncthreads();
    
    local_list_device[((*no_of_topics_device)*node_index)+last_change]=*current_time_device;
    
    __syncthreads();
    
 
   //Update with new weights
   //bookkeeping of no of copies of topics
   
   clock_t end_parallel_device= clock();
   
   if(index==0)
    {
         n=NO_OF_USER*(*no_of_topics_device);
       
        
         for(count=0;count<*no_of_topics_device;count++)
         {
            result_device[count]=0;
         }
          
         
         
         for(count=0;count<n;count++)
         {
            user=count/(*no_of_topics_device);
            topic=count%(*no_of_topics_device);
            if(active_now_device[user]==1)
            {
                if(local_list_device[count] > -1000.0f)
                result_device[topic]++;
            }
            else
            {
                if(local_list_device[count] > -1000.0f)
                    
                {
                
                     weight=INITIAL_WEIGHT_LL*exp(-1*LOCAL_DECAY_RATE *  (*current_time_device-local_list_device[count]));
                    if(weight<WEIGHT_THRESOLD)
                     local_list_device[count]=-9999.0f;
                     else
                     {
                          result_device[topic]++;
                    }
                }
            }
            
         }
    }
    else
    	return;

 
 
             
}
    
    




/*
      **********************************************
      **                                          **
      **       Serial execution Function          **
      **                                          **
      ********************************************** 
 */  
   








void Take_Action_serial(int *v_graph,int *e_graph ,float*local_list,float*global_list,int no_of_topics,float current_time,int *result,int user,short * active_now)
{

	
	//printf("user :%d\n\n",user);
	float weights[800];
    float roulette_wheel[800];
    int current_index,count=0,i,j;
    float weight,sum=0.0f,x=0.0f;
    int neighbours,last_change,N;
    
    if(active_now[user]==0)
    	return;
    
    
    neighbours=v_graph[user+1]-v_graph[user];
    for(i=0;i<no_of_topics;i++)
        weights[i]=0.0f;
        
     for(i=0;i<neighbours;i++)
    {
        current_index=v_graph[user]+i;
        j=(no_of_topics)*e_graph[current_index];
        N=no_of_topics+j;
        for(count=j;count<N;count++)
        {
            if(local_list[count] < -1000.0f)
                continue;
            weight=INITIAL_WEIGHT_LL*(float)exp(-1*LOCAL_DECAY_RATE * (current_time-local_list[count]));
            if(weight<WEIGHT_THRESOLD)
                local_list[count]=-9999.0f;
            else
            {
               weights[count-j]=weights[count-j]+weight;
               sum=sum+weight;
            }
               
                
        }
    }


	 for(count=0;count<no_of_topics;count++)
    {
         if(global_list[count]>current_time)
  			break;
        if(global_list[count] < -1000.0f)
                continue;
        weight=INITIAL_WEIGHT_GL*(float)exp(-1*GLOBAL_DECAY_RATE * (current_time-global_list[count]));
        if(weight>=WEIGHT_THRESOLD)
            {
               weights[count]=weights[count]+weight;
               sum=sum+weight;
            }
    }
	

	roulette_wheel[0]=0.0f;
    x=0.0f;
     for(count=0;count<no_of_topics;count++)
     {
         weights[count]=weights[count]/sum;
        roulette_wheel[count+1]=x+ weights[count];
        x=roulette_wheel[count+1]; 
     }
     
     x=(float)rand()/(float)RAND_MAX;
     //printf("\n\nx=%f\n\n",x);
     last_change=0;
     i=0;
     
    while( roulette_wheel[i]<x)
    {
        if(i>0 && roulette_wheel[i]!=roulette_wheel[i-1])
        	last_change=i;
        i++;
       }
     i=last_change;
     
    local_list[((no_of_topics)*user)+last_change]=current_time;
    
 }
    





/*
      **********************************************
      **                                          **
      **             Main function                **
      **                                          **
      ********************************************** 
 */  
   







int main(void)
{
    
 
    
    clock_t start_main = clock();
    
    int *v_graph ,*e_graph,*result;
    float *local_list,*global_list;
    short  *active_now,*active_element_ids,count;
    float current_inteval_start,current_interval_end,current_time;
    int i,j,a,topic;
    unsigned long long int n;
    int **topic_book_keeping;
    
    float **user_action,weight;
    int *user_current_state;
    
    hiprandState* devStates;
    
    
    size_t array_size,block_size,num_blocks; 
    float *local_list_device,*global_list_device;
    int *v_graph_device ,*e_graph_device,*result_device;
    short  *active_element_ids_device, *count_device,* active_now_device;
    float *current_time_device;
    int *no_of_topics_device,*zero_array;
   // uint kernelTime;
    
    
    double common_time;
    double parallel_time=0.0f;
    double serial_time;
    double total_parallel;
    double total_serial;
    double speed_up;
    double speed_up2;
    
  
    
    
    v_graph=(int*)malloc((NO_OF_USER+1)*sizeof(int));
    e_graph=(int*)malloc(NO_OF_USER*10*sizeof(int));
    
    global_list=(float*)malloc(SIZE_OF_GLOBAL_LIST*sizeof(float));
    active_now=(short *)malloc(NO_OF_USER*sizeof(short));
    


    
    
    
    
    
   make_graph(v_graph,e_graph);
   make_global_list(global_list);
   
   
   no_of_iteration=MAX_TIMESTAMP/SAMPLING_INTERVAL;
   
    user_action=(float**)malloc(NO_OF_USER*sizeof(float*));
    for(j=0;j<NO_OF_USER;j++)
		 user_action[j]=(float*)malloc((no_of_iteration+1)*sizeof(float)); 
	
    
    
    user_current_state=(int*)calloc(NO_OF_USER,sizeof(int)); 
   
   result=(int *)malloc(no_of_topics*sizeof(int));
   for(j=0;j<no_of_topics;j++)
		result[j]=0; 
	
	zero_array= (int *)calloc(no_of_topics,sizeof(int)); 
		
		
   topic_book_keeping=(int **)calloc(no_of_topics,sizeof(int*));
   for(i=0;i<no_of_topics;i++)
        topic_book_keeping[i]=(int *)calloc((no_of_iteration+1),sizeof(int));
   
   local_list=(float*)malloc(NO_OF_USER*no_of_topics*sizeof(float));
   
   initialize_graph(global_list,local_list,topic_book_keeping);
  
  
   generate_initial_action_time(user_action);
 
 

    
    
    array_size=(NO_OF_USER+1)*sizeof(int);
    hipMalloc((void **) &v_graph_device,array_size);
    hipMemcpy(v_graph_device,v_graph,array_size, hipMemcpyHostToDevice);
    
    array_size=NO_OF_USER*10*sizeof(int);
    hipMalloc((void **) &e_graph_device,array_size);
    hipMemcpy(e_graph_device,e_graph,array_size, hipMemcpyHostToDevice);
    
    array_size=NO_OF_USER*no_of_topics*sizeof(float);
    hipMalloc((void **) &local_list_device,array_size);
    hipMemcpy(local_list_device,local_list,array_size, hipMemcpyHostToDevice);
    
    array_size=SIZE_OF_GLOBAL_LIST*sizeof(float);
    hipMalloc((void **) &global_list_device,array_size);
    hipMemcpy(global_list_device,global_list,array_size, hipMemcpyHostToDevice);
    
       
    array_size=sizeof(int);
    hipMalloc((void **) &no_of_topics_device,array_size);
    hipMemcpy(no_of_topics_device,&no_of_topics,array_size,hipMemcpyHostToDevice);
    
   
    array_size=NO_OF_USER*sizeof(short int);
    hipMalloc((void **) &active_now_device,array_size);
    
    array_size=sizeof(float);
    hipMalloc((void **) &current_time_device,array_size);
    
    array_size=no_of_topics*sizeof(int);
    hipMalloc((void **) &result_device,array_size);
    
    
    
     printf("\n\nInitial status :"); 
           for(j=0;j<no_of_topics;j++)
           		printf("%d,",topic_book_keeping[j][0]); 
   
   
   
   
    clock_t end_main= clock();
    
    
    
    
    
    
    
    
    /*
      **********************************************
      **                                          **
      **        Parallel execution starts         **
      **                                          **
      ********************************************** 
  */  
   
   
   
   
 
    for(i=0;i<no_of_iteration;i++)
    {
        count=0;
        current_inteval_start=SAMPLING_INTERVAL*i;
        current_interval_end=SAMPLING_INTERVAL*(i+1);
        current_time=current_inteval_start+(SAMPLING_INTERVAL/2);
        
        
         printf("\n\n Current time interval : %lf-%lf",current_inteval_start,current_interval_end);
    
        for(j=0;j<NO_OF_USER;j++)
        {
            if(user_action[j][user_current_state[j]]>=current_inteval_start && user_action[j][user_current_state[j]]<current_interval_end)
            {
                active_now[j]=1;
                count++;
               user_current_state[j]++;
                
            }
            else
            {
                while(user_action[j][user_current_state[j]]<current_inteval_start)
                	 user_current_state[j]++;
                active_now[j]=0;
                
              }
        }
        
        active_element_ids=(short*)malloc(count*sizeof(short));
      
        a=0;
        for(j=0;j<NO_OF_USER;j++)
        {
            if(active_now[j]==1)
            {
                  active_element_ids[a]=j;
                  a++;
            }
        }
        
        printf("\n No of active users :%d",count );
       
        array_size=count*sizeof(short);
        hipMemcpy(active_now_device,active_now,array_size, hipMemcpyHostToDevice);
        
        hipMalloc((void **) &active_element_ids_device,array_size);
        hipMemcpy(active_element_ids_device,active_element_ids,array_size,hipMemcpyHostToDevice);
        
        array_size=sizeof(short);
        hipMalloc((void **) &count_device,array_size);
        hipMemcpy(count_device,&count,array_size, hipMemcpyHostToDevice);
        
        
         array_size=sizeof(float);

        hipMemcpy(current_time_device,&current_time,array_size,hipMemcpyHostToDevice);
        
        
        hipMalloc ( &devStates, count*sizeof( hiprandState ) );
        
    
      array_size=no_of_topics*sizeof(int);
      hipMemcpy(result_device,zero_array,array_size,hipMemcpyHostToDevice);
     
     
       block_size=count;
       num_blocks=1;
       
       dim3 dimBlock(block_size);
       dim3 dimGrid(num_blocks);
       
     clock_t start_parallel= clock();
       
      //  cutCreateTimer(&kernelTime);
 		//cutResetTimer(kernelTime);
       
     Take_Action_Kernel<<<dimGrid,dimBlock>>>(v_graph_device,e_graph_device,local_list_device,global_list_device,
                                               active_now_device,active_element_ids_device,count_device,no_of_topics_device,
                                                current_time_device,result_device,devStates,unsigned(time(NULL)));
                   
                   
       hipDeviceSynchronize();                                         
      // cutStopTimer(kernelTime);                      
       clock_t end_parallel = clock();
       
       parallel_time=(double)(end_parallel - start_parallel) / CLOCKS_PER_SEC;
       
        array_size=no_of_topics*sizeof(int);
        hipMemcpy(result, result_device,array_size,hipMemcpyDeviceToHost);
        
        
 
         
  
            
        for(j=0;j<no_of_topics;j++)
            topic_book_keeping[j][i+1]=result[j];
         
    printf("\n\nfor iteration %d :",i+1); 
           for(j=0;j<no_of_topics;j++)
  				printf("%d,",result[j]); 
      
        
         free(active_element_ids);
         hipFree(active_element_ids_device);
         
    
 }
     
   
  
  
  clock_t start_serial = clock(); 
  
  
  
  
/*
      **********************************************
      **                                          **
      **         Serial execution starts          **
      **                                          **
      ********************************************** 
 */  
   
 
 
 
 
 initialize_graph(global_list,local_list,topic_book_keeping);
 make_global_list(global_list);
 generate_initial_action_time(user_action);
 
 
   printf("\n\nInitial status :"); 
           for(j=0;j<no_of_topics;j++)
           		printf("%d,",topic_book_keeping[j][0]); 
      
 
  
 
    for(i=0;i<no_of_iteration;i++)
    {
    	
  		count=0;
        current_inteval_start=SAMPLING_INTERVAL*i;
        current_interval_end=SAMPLING_INTERVAL*(i+1);
        current_time=current_inteval_start+(SAMPLING_INTERVAL/2);
        
        
         printf("\n\n Current time interval : %lf-%lf",current_inteval_start,current_interval_end);
    
        for(j=0;j<NO_OF_USER;j++)
        {
            if(user_action[j][user_current_state[j]]>=current_inteval_start && user_action[j][user_current_state[j]]<current_interval_end)
            {
                active_now[j]=1;
                count++;
               user_current_state[j]++;
                
            }
            else
            {
                while(user_action[j][user_current_state[j]]<current_inteval_start)
                	 user_current_state[j]++;
                active_now[j]=0;
                
              }
        }
        
       
    	 count=0;    
          for(j=0;j<NO_OF_USER;j++)
        {
            if(active_now[j]==1)
            {
                count++;
            }
        }
        
        printf("\n No of active users :%d",count );
        
        
     
   
  		
  		for(j=0;j<NO_OF_USER;j++)
  		{
  			 	Take_Action_serial(v_graph,e_graph ,local_list,global_list,no_of_topics,current_time,result,j,active_now);
 
  		}
  		
  
  	
  		n=NO_OF_USER*no_of_topics;
       
        
         for(count=0;count<no_of_topics;count++)
         {
            result[count]=0;
         }
          
  
   
          for(count=0;count<n;count++)
         {
           // user=count/(no_of_topics);
            topic=count%(no_of_topics);
            // printf("topics=%d\n\n",topic);
            if(local_list[count] > -1000.0f)
                    
                {
                
                     weight=INITIAL_WEIGHT_LL*exp(-1*LOCAL_DECAY_RATE *  (current_time-local_list[count]));
                    if(weight<WEIGHT_THRESOLD)
                     local_list[count]=-9999.0f;
                     else
                     {
                          result[topic]++;
                    }
                }
            
            
         }
         
       //  printf("I m here\n\n");
         
         
     for(count=0;count<no_of_topics;count++)
    {
         if(global_list[count]>current_time)
  			break;
          
          if(global_list[count] < -1000.0f)
                continue;
        weight=INITIAL_WEIGHT_GL*(float)exp(-1*GLOBAL_DECAY_RATE * (current_time-global_list[count]));
        if(weight<WEIGHT_THRESOLD)
          global_list[count]=-9999.0f;
    }
         
         
         
          for(j=0;j<no_of_topics;j++)
            topic_book_keeping[j][i+1]=result[j];
         
    	printf("\n\nfor iteration %d :",i+1); 
        for(j=0;j<no_of_topics;j++)
  			printf("%d,",result[j]); 
  			
 
  	
  	}
  
  
  
  
 clock_t end_serial = clock(); 
  
  
  

  
  common_time=(double)(end_main - start_main) / CLOCKS_PER_SEC;
  //parallel_time=(double)(end_parallel - start_parallel) / CLOCKS_PER_SEC;
  serial_time=(double)(end_serial - start_serial) / CLOCKS_PER_SEC;
  total_parallel=common_time+parallel_time;
  total_serial=common_time+serial_time;
  speed_up=total_serial/total_parallel;
  speed_up2=serial_time/parallel_time;
  
  
    
 /*
  printf("\n\nCommon time Elapsed: %f seconds\n",  common_time);  
  printf("\n\nParallel time Elapsed: %f seconds\n", parallel_time);
  printf("\n\nSerial time Elapsed: %f seconds\n",  serial_time);
  printf("\n\nTotal time Elapsed(Palallel execution): %f seconds\n",total_parallel);
  printf("\n\nTotal time Elapsed(serial execution): %f seconds\n",total_serial);
  printf("\n\nSpeed up :%f\n\n",speed_up);
  printf("\n\nSpeed up2 :%f\n\n",speed_up2);
  // printf ("Time for the kernel: %f ms\n", cutGetTimerValue(kernelTime));
  
  */
 
   free(v_graph);
   free(e_graph);
   free(local_list);
   free(global_list);
   free(active_now);
   free(result);
   free(zero_array);
   
   
   hipFree(v_graph_device);
   hipFree(e_graph_device);
   hipFree(local_list_device);
   hipFree(global_list_device);
   hipFree(no_of_topics_device);
   hipFree(current_time_device);
   hipFree(active_now_device);
   hipFree(result_device);
 
  
   
    
}





/*
      **********************************************
      **                                          **
      **             Graph creation               **
      **                                          **
      ********************************************** 
 */  
   





void make_graph(int *v_graph , int *e_graph)
{
 
 map<int,set<int> > graph;
 map<int,set<int> >::iterator map_itr;
 set<int>::iterator set_itr;
 set<int> nullset;
 set<int> hash;
 set<int> :: iterator itr;
 int no,i,neighbour,count,max,min;
 float r;
 int *neighbours;
 int edge_index;
 
 
 
 neighbours=(int*)calloc(NO_OF_USER,sizeof(int));
 

 
 for(i=0;i<NO_OF_USER;i++)
 	graph.insert(std::pair<int,set<int> >(i,nullset));

 for(i=0;i<NO_OF_USER;i++)
 {
    max=MAX_NEIGHBOUR-neighbours[i];
    min=MIN_NEIGHBOUR-neighbours[i];
    if(min<0)
     min=0;
     if(max>0)
     {
       r=(float)rand()/(float)(RAND_MAX);
       no=(int)(((float)(max-min)*r)+min);
       count=0;
       while(count<no)
        {
	    	srand(rand());
            neighbour=(int)(rand() % NO_OF_USER) ;
	        set_itr =graph[i].find(neighbour);
            if(neighbour==i || set_itr!=graph[i].end())
                continue;
	      
	     graph[i].insert(neighbour);
	     graph[neighbour].insert(i);
	     neighbours[i]++;
	     neighbours[neighbour]++;
	     count++;
		}
     }
 }
 
 i=0;
 edge_index=0;
 
 for (map_itr=graph.begin(); map_itr!=graph.end(); ++map_itr)
 {
   v_graph[i]=edge_index;
   
   for (set_itr=graph[i].begin(); set_itr!=graph[i].end(); ++set_itr)
   {
   	  e_graph[edge_index]=*set_itr;
   	  edge_index++;
   }
   map_itr->second.clear();
   i++;
 }
       
 graph.clear();
 
}






/*
      **********************************************
      **                                          **
      **              Global list                 **
      **                                          **
      ********************************************** 
 */  
   




void make_global_list(float *global_list)
{
    float t=0.0f,r=0.0f;
    int i=0,n,max;
    n=SIZE_OF_GLOBAL_LIST;
    max=TIMEUNIT_BEFORE_ZERO+MAX_TIMESTAMP;
      
      while (i<n && t<max)
      {
       //srand(rand());
       r=(float)rand()/(float)(RAND_MAX);
       t=t-(log(r)/LAMDA_ARRIVAL_GLOBAL);
        global_list[i]=t-TIMEUNIT_BEFORE_ZERO;
        if(global_list[i-1]<0 && global_list[i]>=0)
            global_zero_index=i;
        i++;
       }
       no_of_topics=i;
       
}





/*
      **********************************************
      **                                          **
      **         Initialization of graph          **
      **                                          **
      ********************************************** 
 */  
   
   
   




void initialize_graph(float *global_list,float *local_list,int **topic_book_keeping)
{
    
    float *weights,*roulette_wheel;
    int no,i,j;
    float sum=0.0f,x=0.0f;
    
    no=NO_OF_USER*no_of_topics;
   
    for(i=0;i<no;i++)
        local_list[i]=-9999.0f;
    
    weights=(float *)malloc(global_zero_index*sizeof(float));
    roulette_wheel=(float *)malloc((global_zero_index+1)*sizeof(float));
    roulette_wheel[0]=0;
    
    for(i=0;i<global_zero_index;i++)
    {
        weights[i]=INITIAL_WEIGHT_GL*exp(GLOBAL_DECAY_RATE * global_list[i]);
        sum=sum+ weights[i];
    }
     
     for(i=0;i<global_zero_index;i++)
    {
        weights[i]=weights[i]/sum;
        roulette_wheel[i+1]=x+ weights[i];
        x=roulette_wheel[i+1];
    }
    
    
   for(j=0;j<NO_OF_USER;j++)
   {
    i=0;
    srand(rand());
    x=(float)rand()/(float)(RAND_MAX);
    while( roulette_wheel[i]<x)
        i++;
    i--;
        
    local_list[(j*no_of_topics)+i]=0.0f;
    topic_book_keeping[i][0]++;
    
    
   }
   
      
   free(weights);
   free(roulette_wheel);
    
    
}




/*
      **********************************************
      **                                          **
      **    Genartion of initial action time      **
      **                                          **
      ********************************************** 
 */  
   




void generate_initial_action_time(float **user_action)
{
    float t=0.0f,r=0.0f;
    int i=0,j;
      
    for(i=0;i<NO_OF_USER;i++)
    {
       t=0.0f;
       r=0.0f;
       
       for(j=0;j<no_of_iteration+1;j++)
       {
        srand(rand());
        r=(float)rand()/(float)(RAND_MAX);
        t=-(log(r)/LAMDA_ARRIVAL_USER);
        if(j==0)
        	user_action[i][j]=t;
        else
        	user_action[i][j]=user_action[i][j-1]+t;
    	}
    }
     
}





 

